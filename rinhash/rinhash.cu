#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <stdexcept>

// Include shared device functions (chỉ include .cuh hoặc .h)
#include "rinhash_device.cuh"
#include "argon2d_device.cuh"
#include "sha3-256.cu"
#include "blake3_device.cuh"

// Số block tối đa cho batch (tùy GPU)
#define MAX_BATCH_BLOCKS 16384

// Kernel đơn: mỗi lần chỉ chạy 1 thread
extern "C" __global__ void rinhash_cuda_kernel(
    const uint8_t* input, 
    size_t input_len, 
    uint8_t* output,
    block* memory,      // bộ nhớ argon2 đã cấp phát trên host, truyền vào
    uint32_t m_cost
) {
    // Chỉ 1 thread xử lý
    if (threadIdx.x == 0) {
        // Step 1: BLAKE3 hash - now using light_hash_device
        light_hash_device(input, input_len, blake3_out);
        // Step 2: Argon2d hash
        uint32_t m_cost = 64; // Example
        size_t memory_size = m_cost * sizeof(block);
        block* d_memory = (block*)malloc(memory_size);
        uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
        uint8_t argon2_out[32];
        device_argon2d_hash(argon2_out, blake3_out, 32, 2, m_cost, 1, memory, salt, sizeof(salt));

        uint8_t sha3_out[32];
        sha3_256_device(argon2_out, 32, sha3_out);
    }
}

// Kernel batch: mỗi thread xử lý 1 block header
extern "C" __global__ void rinhash_cuda_kernel_batch(
    const uint8_t* headers,         // num_blocks * 80 bytes
    size_t header_len,              // = 80
    uint8_t* outputs,               // num_blocks * 32 bytes
    uint32_t num_blocks,
    block* memories,                // num_blocks * m_cost * sizeof(block)
    uint32_t m_cost
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_blocks) return;
    const uint8_t* input = headers + tid * header_len;
    uint8_t* output = outputs + tid * 32;
    block* memory = memories + tid * m_cost;

    uint8_t blake3_out[32];
    light_hash_device(input, header_len, blake3_out);

    uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
    uint8_t argon2_out[32];
    device_argon2d_hash(argon2_out, blake3_out, 32, 2, m_cost, 1, memory, salt, sizeof(salt));

    sha3_256_device(argon2_out, 32, output);
}


// Helper: kiểm tra lỗi CUDA
inline void check_cuda(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s: %s\n", msg, hipGetErrorString(err));
        throw std::runtime_error("CUDA error");
    }
}

// RinHash CUDA implementation (single)
extern "C" void rinhash_cuda(const uint8_t* input, size_t input_len, uint8_t* output) {
    uint8_t *d_input = nullptr;
    uint8_t *d_output = nullptr;
    block* d_memory = nullptr;
    uint32_t m_cost = 64;

    hipError_t err;

    // Alloc device memory
    err = hipMalloc(&d_input, input_len);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc input fail\n"); return; }

    err = hipMalloc(&d_output, 32);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc output fail\n"); hipFree(d_input); return; }

    err = hipMalloc(&d_memory, m_cost * sizeof(block));
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc argon2 memory fail\n"); hipFree(d_input); hipFree(d_output); return; }

    // Copy input
    err = hipMemcpy(d_input, input, input_len, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy input fail\n"); hipFree(d_input); hipFree(d_output); hipFree(d_memory); return; }

    // Launch kernel
    rinhash_cuda_kernel<<<256, 1024>>>(d_input, input_len, d_output, d_memory, m_cost);
    hipDeviceSynchronize();
    check_cuda("rinhash_cuda_kernel");

    // Copy result
    err = hipMemcpy(output, d_output, 32, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy output fail\n"); }

    // Free
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_memory);
}

// Batch processing version for mining
extern "C" void rinhash_cuda_batch(
    const uint8_t* block_headers,
    size_t block_header_len,
    uint8_t* outputs,
    uint32_t num_blocks
) {
    if (num_blocks > MAX_BATCH_BLOCKS) {
        fprintf(stderr, "Batch too large (max %u)\n", MAX_BATCH_BLOCKS);
        return;
    }

    uint8_t *d_headers = nullptr, *d_outputs = nullptr;
    block* d_memories = nullptr;
    uint32_t m_cost = 64;
    size_t headers_size = block_header_len * num_blocks;
    size_t outputs_size = 32 * num_blocks;
    size_t memories_size = num_blocks * m_cost * sizeof(block);

    hipError_t err;
    err = hipMalloc(&d_headers, headers_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc headers fail\n"); return; }
    err = hipMalloc(&d_outputs, outputs_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc outputs fail\n"); hipFree(d_headers); return; }
    err = hipMalloc(&d_memories, memories_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc argon2 memories fail\n"); hipFree(d_headers); hipFree(d_outputs); return; }

    hipMemset(d_outputs, 0xee, outputs_size);
    hipMemcpy(d_headers, block_headers, headers_size, hipMemcpyHostToDevice);

    const int threads_per_block = 128;
    int blocks = (num_blocks + threads_per_block - 1) / threads_per_block;
    rinhash_cuda_kernel_batch<<<blocks, threads_per_block>>>(
        d_headers, block_header_len, d_outputs, num_blocks, d_memories, m_cost
    );
    hipDeviceSynchronize();
    check_cuda("rinhash_cuda_kernel_batch");

    err = hipMemcpy(outputs, d_outputs, outputs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy output batch fail\n"); }

    hipFree(d_headers);
    hipFree(d_outputs);
    hipFree(d_memories);
}

// Helper function to convert a block header to bytes
extern "C" void blockheader_to_bytes(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output,
    size_t* output_len
) {
    size_t offset = 0;
    memcpy(output + offset, version, 4); offset += 4;
    memcpy(output + offset, prev_block, 32); offset += 32;
    memcpy(output + offset, merkle_root, 32); offset += 32;
    memcpy(output + offset, timestamp, 4); offset += 4;
    memcpy(output + offset, bits, 4); offset += 4;
    memcpy(output + offset, nonce, 4); offset += 4;
    *output_len = offset;
}

// Main RinHash function that would be called from outside
extern "C" void RinHash(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output
) {
    uint8_t block_header[80]; // Standard block header size
    size_t block_header_len;
    blockheader_to_bytes(
        version,
        prev_block,
        merkle_root,
        timestamp,
        bits,
        nonce,
        block_header,
        &block_header_len
    );
    rinhash_cuda(block_header, block_header_len, output);
}

bool is_better(uint8_t* hash1, uint8_t* hash2) {
    for (int i = 7; i >= 0; i--) {
        uint32_t h1 = ((uint32_t)hash1[i*4 + 0]) |
                      ((uint32_t)hash1[i*4 + 1] << 8) |
                      ((uint32_t)hash1[i*4 + 2] << 16) |
                      ((uint32_t)hash1[i*4 + 3] << 24);
        uint32_t h2 = ((uint32_t)hash2[i*4 + 0]) |
                      ((uint32_t)hash2[i*4 + 1] << 8) |
                      ((uint32_t)hash2[i*4 + 2] << 16) |
                      ((uint32_t)hash2[i*4 + 3] << 24);
        if (h1 < h2) return true;
        if (h1 > h2) return false;
    }
    return false; // equal
}

// Mining function that tries different nonces
extern "C" void RinHash_mine(
    const uint32_t* work_data,
    uint32_t nonce_offset,
    uint32_t start_nonce,
    uint32_t num_nonces,
    uint32_t* found_nonce,
    uint8_t* target_hash,
    uint8_t* best_hash
) {
    const size_t block_header_len = 80;
    int headerbytes = block_header_len * num_nonces;
    int hashbytes = 32 * num_nonces;
    uint8_t block_headers[80 * 1024];
    uint8_t hashes[32 * 1024];
    hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024); // 128MB
    // Prepare block headers with different nonces
    for (uint32_t i = 0; i < num_nonces; i++) {
        uint32_t current_nonce = start_nonce + i;
        uint32_t work_data_copy[20];
        memcpy(work_data_copy, work_data, 80);
        work_data_copy[nonce_offset] = current_nonce;
        memcpy(&block_headers[i * block_header_len], work_data_copy, 80);
    }

    // Calculate hashes for all nonces
    rinhash_cuda_batch(block_headers.data(), block_header_len, hashes.data(), num_nonces);

    // Initialize best_hash with the first hash
    memcpy(best_hash, hashes.data(), 32);
    *found_nonce = start_nonce;
    for (uint32_t i = 1; i < num_nonces; i++) {
        uint8_t* current_hash = hashes.data() + i * 32;
        if (is_better(current_hash, best_hash)) {
            memcpy(best_hash, current_hash, 32);
            *found_nonce = start_nonce + i;
        }
    }
}
